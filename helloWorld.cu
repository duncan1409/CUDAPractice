
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel()
{
	printf("Hello World!! in GPU\n");
}
int main()
{
	kernel<<<1, 1>>>();
	printf("Hello World!! in CPU\n");
	return 0;
}