
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <random>

using namespace std;

__global__ void addKernel(int *a, int *b, int *c, int *d)
{
	int i = threadIdx.x;
	d[i] = a[i] + b[i] + c[i];
}
int main()
{
	const int SIZE = 5;
	int a[SIZE] = {0};
	int b[SIZE] = {0};
	int c[SIZE] = {0};
	int d[SIZE] = {0};

	// input random number in array
	random_device rd;
	mt19937 gen(rd());
	uniform_int_distribution<int> dis(0, 99);

	for (int i = 0; i < SIZE; i++)
	{
		a[i] = dis(gen);
		b[i] = dis(gen);
		c[i] = dis(gen);
	}

	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	int *dev_d = 0;

	hipMalloc((void **)&dev_a, SIZE * sizeof(int));
	hipMalloc((void **)&dev_b, SIZE * sizeof(int));
	hipMalloc((void **)&dev_c, SIZE * sizeof(int));
	hipMalloc((void **)&dev_d, SIZE * sizeof(int));

	hipMemcpy(dev_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, SIZE * sizeof(int), hipMemcpyHostToDevice);
	addKernel<<<1, SIZE>>>(dev_a, dev_b, dev_c, dev_d);
	hipMemcpy(d, dev_d, SIZE * sizeof(int), hipMemcpyDeviceToHost);

	printf("{%d, %d, %d, %d, %d} + {%d, %d, %d, %d, %d} + {%d, %d, %d, %d, %d} = {%d, %d, %d, %d, %d}\n",
		   a[0], a[1], a[2], a[3], a[4], b[0], b[1], b[2], b[3], b[4], c[0], c[1], c[2], c[3], c[4], d[0], d[1], d[2], d[3], d[4]);

	hipFree(dev_d);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}