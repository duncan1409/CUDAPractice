
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void addKernel(int *a, int *b, int *c)
{

	int x = threadIdx.x;
	int y = threadIdx.y;
	int i = y * (blockDim.x) + x; // index = y * WIDTH + x
	c[i] = a[i] + b[i];
}

int main()
{
	const int WIDTH = 5;
	int a[WIDTH][WIDTH];
	int b[WIDTH][WIDTH];
	int c[WIDTH][WIDTH] = {0};

	// Host에서 배열 'a'와 'b'를 채운다.
	for (int y = 0; y < WIDTH; y++)
	{
		for (int x = 0; x < WIDTH; x++)
		{
			a[y][x] = y * 10 + x;
			b[y][x] = (y * 10 + x) * 100;
		}
	}

	int *dev_a, *dev_b, *dev_c = 0; // GPU does not know the array structure of dev_a, dev_b, dev_c

	hipMalloc((void **)&dev_a, WIDTH * WIDTH * sizeof(int));
	hipMalloc((void **)&dev_b, WIDTH * WIDTH * sizeof(int));
	hipMalloc((void **)&dev_c, WIDTH * WIDTH * sizeof(int));

	hipMemcpy(dev_a, a, WIDTH * WIDTH * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, WIDTH * WIDTH * sizeof(int), hipMemcpyHostToDevice);

	dim3 DimBlock(WIDTH, WIDTH);
	addKernel<<<1, DimBlock>>>(dev_a, dev_b, dev_c);

	// 배열 'c'를 Device에서 다시 Host로 복사
	hipMemcpy(c, dev_c, WIDTH * WIDTH * sizeof(int), hipMemcpyDeviceToHost);

	for (int y = 0; y < WIDTH; y++)
	{
		for (int x = 0; x < WIDTH; x++)
		{
			printf("%5d", c[y][x]);
		}
		printf("\n");
	}

	return 0;
}