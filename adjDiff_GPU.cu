#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h> //for rand(), malloc(), free()
#include <sys/stat.h>
#include <windows.h>

#define GRIDSIZE 8 * 1024				 // 8K
#define BLOCKSIZE 1024					 // 1K
#define TOTALSIZE (GRIDSIZE * BLOCKSIZE) // 8M

__global__ void adjDiff(float *result, float *input)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i > 0)
	{
		float x_i = input[i];
		float x_i_m1 = input[i - 1];
		result[i] = x_i - x_i_m1;
	}
}
void genData(float *ptr, unsigned int size)
{
	while (size--)
		*ptr++ = (float)(rand() % 1000) / 1000.0F;
}
int main()
{
	float *pSource = NULL;
	float *pResult = NULL;
	int i;
	long long cntStart, cntEnd, freq = 0LL;
	QueryPerformanceFrequency((LARGE_INTEGER *)(&freq));
	pSource = (float *)malloc(TOTALSIZE * sizeof(float));
	pResult = (float *)malloc(TOTALSIZE * sizeof(float));

	// generate input source data
	genData(pSource, TOTALSIZE);

	float *pResultDev = NULL;
	float *pSourceDev = NULL;
	// calculate the adjacent difference
	pResult[0] = 0.0F;
	hipMalloc((void **)&pSourceDev, TOTALSIZE * sizeof(float));
	hipMalloc((void **)&pResultDev, TOTALSIZE * sizeof(float));
	//CUDA mem cpy from host to device
	hipMemcpy(pSourceDev, pSource, TOTALSIZE * sizeof(float), hipMemcpyHostToDevice);
	// start the timer
	QueryPerformanceCounter((LARGE_INTEGER *)(&cntStart));
	//CUDA launch the kernel adjDiff
	dim3 dimGrid(GRIDSIZE, 1, 1);
	dim3 dimBlock(BLOCKSIZE, 1, 1);
	adjDiff<<<dimGrid, dimBlock>>>(pResultDev, pSourceDev);
	QueryPerformanceCounter((LARGE_INTEGER *)(&cntEnd));
	//CUDA memcpy from device to host
	hipMemcpy(pResult, pResultDev, TOTALSIZE * sizeof(float), hipMemcpyDeviceToHost);
	printf("elapsed time = %f usec\n", (double)(cntEnd - cntStart) * 1000000.0 / (double)(freq));

	// print sample cases
	i = 1;
	printf("i = %7d: %f = %f - %f\n", i, pResult[i], pSource[i], pSource[i - 1]);
	i = TOTALSIZE - 1;
	printf("i = %7d: %f = %f - %f\n", i, pResult[i], pSource[i], pSource[i - 1]);
	i = TOTALSIZE / 2;
	printf("i = %7d: %f = %f - %f\n", i, pResult[i], pSource[i], pSource[i - 1]);

	//	free the memory
	free(pResult);
	free(pSource);
}